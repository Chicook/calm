// Based on NVIDIA's cupti_trace_injection sample
#include <assert.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <cupti.h>

#define CUPTI_CHECK(call)                                            \
	do {                                                             \
		CUptiResult _status = call;                                  \
		if (_status != CUPTI_SUCCESS) {                              \
			const char* err = "?";                                   \
			cuptiGetResultString(_status, &err);                     \
			fprintf(stderr, "CUPTI error in %s at %s:%d: %s (%d)\n", \
			        __FUNCTION__, __FILE__, __LINE__, err, _status); \
			abort();                                                 \
		}                                                            \
	} while (0)

#define BUFFER_SIZE 8 * 1024 * 1024
#define MAX_KERNELS 1024

struct KernelInfo {
	const char* name;

	float total_time;
	int calls;
	float call_avg;
	float call_m2;
};

static KernelInfo kernels[MAX_KERNELS];
static int n_kernels;

static KernelInfo* get_kernel(const char* name) {
	for (int i = 0; i < n_kernels; i++) {
		if (strcmp(kernels[i].name, name) == 0) {
			return &kernels[i];
		}
	}

	assert(n_kernels < MAX_KERNELS);
	KernelInfo* kernel = &kernels[n_kernels++];
	kernel->name = name;

	return kernel;
}

static void CUPTIAPI buffer_requested(uint8_t** buffer, size_t* size, size_t* maxNumRecords) {
	*size = BUFFER_SIZE;
	*buffer = (uint8_t*)malloc(BUFFER_SIZE);
	*maxNumRecords = 0;
}

static void CUPTIAPI buffer_completed(hipCtx_t ctx, uint32_t streamId, uint8_t* buffer, size_t size, size_t validSize) {
	CUpti_Activity* record = NULL;

	for (;;) {
		CUptiResult status = cuptiActivityGetNextRecord(buffer, validSize, &record);
		if (status == CUPTI_ERROR_MAX_LIMIT_REACHED) {
			break;
		}
		CUPTI_CHECK(status);

		switch (record->kind) {
		case CUPTI_ACTIVITY_KIND_KERNEL:
		case CUPTI_ACTIVITY_KIND_CONCURRENT_KERNEL: {
			CUpti_ActivityKernel8* activity = (CUpti_ActivityKernel8*)record;
			KernelInfo* info = get_kernel(activity->name);

			float time = (float)(activity->end - activity->start) / 1e6;

			info->total_time += time;

			// Welford's algorithm
			float delta = time - info->call_avg;
			info->calls++;
			info->call_avg += delta / info->calls;
			info->call_m2 += delta * (time - info->call_avg);
			break;
		}
		default:
			break;
		}
	}

	free(buffer);

	size_t dropped = 0;
	CUPTI_CHECK(cuptiActivityGetNumDroppedRecords(ctx, streamId, &dropped));

	if (dropped != 0) {
		printf("WARNING: dropped %u CUPTI activity records.\n", (unsigned int)dropped);
	}
}

static void atexit_handler(void) {
	CUPTI_CHECK(cuptiActivityFlushAll(CUPTI_ACTIVITY_FLAG_FLUSH_FORCED));

	if (n_kernels) {
		printf("\n");
		printf("%20s%15s%20s%15s\n", "Kernel", "Time (%)", "Avg Time (us)", "Calls");
		printf("%20s%15s%20s%15s\n", "---", "---", "---", "---");

		float total_time = 0;
		for (int i = 0; i < n_kernels; i++) {
			total_time += kernels[i].total_time;
		}

		for (int i = 0; i < n_kernels; i++) {
			KernelInfo* kernel = &kernels[i];

			const char* name = kernel->name;
			size_t length = strlen(name);

			if (strncmp(name, "_Z", 2) == 0 && length >= 2) {
				name += 2;
				char* end;
				length = strtoul(name, &end, 10);
				name = end;
				length = length > strlen(name) ? strlen(name) : length;
			}

			if (strncmp(name, "kernel_", 7) == 0 && length >= 7) {
				name += 7;
				length -= 7;
			}

			char avgtime[64];
			snprintf(avgtime, sizeof(avgtime), "%.2f ± %.2f",
			         kernel->call_avg * 1e3,
			         sqrtf(kernel->call_m2 / kernel->calls) * 1e3);

			printf("%20.*s%14.1f%%%21s%15d\n", (int)length, name,
			       kernel->total_time / total_time * 100, avgtime, kernel->calls);
		}
	}
}

extern "C" int InitializeInjection(void) {
	atexit(&atexit_handler);

	const char* sync = getenv("PROF_SYNC");

	// note: KIND_KERNEL serializes kernel launches; KIND_CONCURRENT_KERNEL does not but it results in less stable timings
	if (sync && atoi(sync)) {
		CUPTI_CHECK(cuptiActivityEnable(CUPTI_ACTIVITY_KIND_KERNEL));
	} else {
		CUPTI_CHECK(cuptiActivityEnable(CUPTI_ACTIVITY_KIND_CONCURRENT_KERNEL));
	}

	CUPTI_CHECK(cuptiActivityRegisterCallbacks(buffer_requested, buffer_completed));
	return 1;
}

int main() {
}
