#include "hip/hip_runtime.h"
#include "model.h"

#include "profiler.h"

#include <assert.h>
#include <math.h>
#include <stdio.h>

#include <hip/hip_fp16.h>

#include "helpers.cuh"

#define CUDA_CHECK(x)                                                                                    \
	do {                                                                                                 \
		hipError_t err = x;                                                                             \
		if (err != hipSuccess) {                                                                        \
			fprintf(stderr, "CUDA error in %s at %s:%d: %s (%s=%d)\n", __FUNCTION__, __FILE__, __LINE__, \
			        hipGetErrorString(err), hipGetErrorName(err), err);                                \
			abort();                                                                                     \
		}                                                                                                \
	} while (0)

#define CUDA_SYNC() CUDA_CHECK(hipDeviceSynchronize())

static void* cuda_devicecopy(void* host, size_t size) {
	void* device = NULL;
	CUDA_CHECK(hipMalloc(&device, size));
	CUDA_CHECK(hipMemcpy(device, host, size, hipMemcpyHostToDevice));
	return device;
}

static void* cuda_devicealloc(size_t size) {
	void* ptr = NULL;
	CUDA_CHECK(hipMalloc(&ptr, size));
	return ptr;
}

static void* cuda_hostalloc(size_t size) {
	void* ptr = NULL;
	CUDA_CHECK(hipHostAlloc(&ptr, size, 0));
	return ptr;
}

extern "C" void prepare_cuda(struct Transformer* transformer) {
	struct Config* config = &transformer->config;
	struct Weights* weights = &transformer->weights;
	struct RunState* state = &transformer->state;

	hipDeviceProp_t devprop = {};
	CUDA_CHECK(hipGetDeviceProperties(&devprop, 0));

	printf("# CUDA: %s, SM %d.%d, %.1f GiB, peak bandwidth %.0f GB/s\n",
	       devprop.name, devprop.major, devprop.minor,
	       (double)devprop.totalGlobalMem / (1024 * 1024 * 1024),
	       (double)devprop.memoryClockRate * (devprop.memoryBusWidth / 8) * 2 / 1e6);

	int dim = config->dim;
	int hidden_dim = config->hidden_dim;
	int kv_dim = (config->dim * config->n_kv_heads) / config->n_heads;

	for (int l = 0; l < config->n_layers; ++l) {
		weights->rms_att_weight[l] = (dtype_t*)cuda_devicecopy(weights->rms_att_weight[l], dim * sizeof(dtype_t));
		weights->rms_ffn_weight[l] = (dtype_t*)cuda_devicecopy(weights->rms_ffn_weight[l], dim * sizeof(dtype_t));

		weights->wq[l] = (dtype_t*)cuda_devicecopy(weights->wq[l], dim * dim * sizeof(dtype_t));
		weights->wk[l] = (dtype_t*)cuda_devicecopy(weights->wk[l], dim * kv_dim * sizeof(dtype_t));
		weights->wv[l] = (dtype_t*)cuda_devicecopy(weights->wv[l], dim * kv_dim * sizeof(dtype_t));
		weights->wo[l] = (dtype_t*)cuda_devicecopy(weights->wo[l], dim * dim * sizeof(dtype_t));

		weights->w1[l] = (dtype_t*)cuda_devicecopy(weights->w1[l], dim * hidden_dim * sizeof(dtype_t));
		weights->w2[l] = (dtype_t*)cuda_devicecopy(weights->w2[l], dim * hidden_dim * sizeof(dtype_t));
		weights->w3[l] = (dtype_t*)cuda_devicecopy(weights->w3[l], dim * hidden_dim * sizeof(dtype_t));
	}

	weights->rms_final_weight = (dtype_t*)cuda_devicecopy(weights->rms_final_weight, dim * sizeof(dtype_t));
	weights->token_embedding_table = (dtype_t*)cuda_devicecopy(weights->token_embedding_table, config->vocab_size * dim * sizeof(dtype_t));
	weights->wcls = (dtype_t*)cuda_devicecopy(weights->wcls, dim * config->vocab_size * sizeof(dtype_t));

	state->x = (float*)cuda_devicealloc(dim * sizeof(float));
	state->xb = (float*)cuda_devicealloc(dim * sizeof(float));
	state->hb = (float*)cuda_devicealloc(hidden_dim * sizeof(float));
	state->q = (float*)cuda_devicealloc(dim * sizeof(float));
	state->k = (float*)cuda_devicealloc(kv_dim * sizeof(float));
	state->v = (float*)cuda_devicealloc(kv_dim * sizeof(float));
	state->att = (float*)cuda_devicealloc(config->n_heads * config->seq_len * sizeof(float));

	state->key_cache = (kvtype_t*)cuda_devicealloc((size_t)config->n_layers * config->seq_len * kv_dim * sizeof(kvtype_t));
	state->value_cache = (kvtype_t*)cuda_devicealloc((size_t)config->n_layers * config->seq_len * kv_dim * sizeof(kvtype_t));

	// logits are going to be read by the host so we just allocate them in host and write to host directly
	state->logits = (float*)cuda_hostalloc(config->vocab_size * sizeof(float));
}

__global__ static void kernel_embed(float* o, dtype_t* weight, int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	assert(i < size);

	o[i] = float(weight[i]);
}

__global__ static void kernel_rmsnorm(float* o, float* x, dtype_t* weight, int size) {
	int i = threadIdx.x;
	int blockSize = blockDim.x;

	// calculate sum of squares (per thread)
	float ss = 0.0f;
	for (int j = i; j < size; j += blockSize) {
		ss += x[j] * x[j];
	}

	// sum across threads in block
	ss = blockreduce_sum(ss);

	// compute scale
	ss /= size;
	ss += 1e-5f;
	ss = 1.0f / sqrtf(ss);

	// normalize and scale
	for (int j = i; j < size; j += blockSize) {
		o[j] = float(weight[j]) * (ss * x[j]);
	}
}

__global__ static void kernel_matmul_cls(float* xout, float* x, dtype_t* w, int n, int d) {
	int i = blockIdx.x;
	assert(i < d);

	float val = matmul_warppar(x, w, i, n);

	if (threadIdx.x == 0) {
		xout[i] = val;
	}
}

__global__ static void kernel_matmul_qkv(float* qout, float* kout, float* vout, float* x, dtype_t* wq, dtype_t* wk, dtype_t* wv, int n, int d, int kvd) {
	int i = blockIdx.x;
	assert(i < d + kvd * 2);

	float* out = i < d ? qout : (i < d + kvd ? kout : vout);
	dtype_t* w = i < d ? wq : (i < d + kvd ? wk : wv);
	int j = i < d ? i : (i < d + kvd ? i - d : i - d - kvd);

	float val = matmul_warppar(x, w, j, n);
	if (threadIdx.x == 0) {
		out[j] = val;
	}
}

__global__ static void kernel_matmul_attn(float* xout, float* x, dtype_t* w, int n, int d) {
	int i = blockIdx.x;
	assert(i < d);

	float val = matmul_warppar(x, w, i, n);

	if (threadIdx.x == 0) {
		// += for residual
		xout[i] += val;
	}
}

__global__ static void kernel_matmul_ffn13(float* xout, float* x, dtype_t* w1, dtype_t* w3, int n, int d) {
	int i = blockIdx.x;
	assert(i < d);

	float v1 = matmul_warppar(x, w1, i, n);
	float v3 = matmul_warppar(x, w3, i, n);

	// silu(x)=x*σ(x), where σ(x) is the logistic sigmoid
	float val = v1;
	val *= 1.0f / (1.0f + expf(-v1));
	val *= v3;

	if (threadIdx.x == 0) {
		xout[i] = val;
	}
}

__global__ static void kernel_matmul_ffn2(float* xout, float* x, dtype_t* w, int n, int d) {
	int i = blockIdx.x;
	assert(i < d);

	float val = matmul_warppar(x, w, i, n);

	if (threadIdx.x == 0) {
		// += for residual
		xout[i] += val;
	}
}

__global__ static void kernel_rope_kv(float* q, float* k, float* v, kvtype_t* kb, kvtype_t* vb, int head_size, int pos, float theta, int d, int kvd, int seq_len) {
	int i = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
	assert(i < d);

	int head_dim = i % head_size;
	float freq = 1.0f / powf(theta, head_dim / (float)head_size);
	float val = pos * freq;
	float fcr = cosf(val);
	float fci = sinf(val);

	float q0 = q[i];
	float q1 = q[i + 1];
	q[i] = q0 * fcr - q1 * fci;
	q[i + 1] = q0 * fci + q1 * fcr;

	if (i < kvd) {
		float k0 = k[i];
		float k1 = k[i + 1];
		float rk0 = k0 * fcr - k1 * fci;
		float rk1 = k0 * fci + k1 * fcr;

		k[i] = rk0;
		k[i + 1] = rk1;

		// update kvcache key/value
		kb[pos * kvd + i] = rk0;
		kb[pos * kvd + i + 1] = rk1;

		// note: v layout is transposed (we store all positions for a given head contiguously) to improve attn_mix performance
		vb[pos + seq_len * i] = v[i];
		vb[pos + seq_len * (i + 1)] = v[i + 1];
	}
}

__global__ static void kernel_attn_score(float* attb, float* qb, kvtype_t* kb, int n_kv_heads, int head_size, int seq_len, int kv_dim, int kv_mul, int pos) {
	int t = blockIdx.x;
	assert(t <= pos);

	int kvh = blockIdx.y;
	assert(kvh < n_kv_heads);

	int h = kvh * kv_mul + threadIdx.y;

	float* q = qb + h * head_size;
	kvtype_t* k = kb + t * kv_dim + kvh * head_size;
	float* att = attb + h * seq_len;

	float score = 0.0f;
	for (int j = threadIdx.x * 2; j < head_size; j += warpSize * 2) {
		float2 kk = __half22float2(*((half2*)&k[j]));
		score += kk.x * q[j];
		score += kk.y * q[j + 1];
	}

	score = warpreduce_sum(score);
	score /= sqrtf(head_size);

	if (threadIdx.x == 0) {
		att[t] = score;
	}
}

__global__ static void kernel_attn_softmax(float* attb, int n_heads, int seq_len, int pos) {
	int i = threadIdx.x;

	int h = blockIdx.x;
	assert(h < n_heads);

	float* att = attb + h * seq_len;

	// find max value per thread (for numerical stability)
	float max_val = 0.f;
	for (int j = i; j <= pos; j += blockDim.x) {
		max_val = max(max_val, att[j]);
	}

	// max across threads in block
	max_val = blockreduce_max(max_val);

	// exp and sum per thread
	float sum = 0.0f;
	for (int j = i; j <= pos; j += blockDim.x) {
		sum += expf(att[j] - max_val);
	}

	// sum across threads in block
	sum = blockreduce_sum(sum);

	// output normalized values
	for (int j = i; j <= pos; j += blockDim.x) {
		att[j] = expf(att[j] - max_val) / sum;
	}
}

__global__ static void kernel_attn_mix(float* xout, float* attb, kvtype_t* valb, int n_kv_heads, int head_size, int seq_len, int kv_dim, int kv_mul, int pos) {
	int i = blockIdx.x;
	assert(i < head_size);

	int kvh = blockIdx.y;
	assert(kvh < n_kv_heads);

	int h = kvh * kv_mul + threadIdx.y;

	float* att = attb + h * seq_len;
	kvtype_t* val = valb + (kvh * head_size + i) * seq_len;

	float res = 0.0f;
	for (int t = threadIdx.x; t <= pos; t += warpSize) {
		res += att[t] * float(val[t]);
	}

	res = warpreduce_sum(res);

	if (threadIdx.x == 0) {
		xout[h * head_size + i] = res;
	}
}

extern "C" float* forward_cuda(struct Transformer* transformer, int token, int pos, unsigned flags) {
	profiler_begin();

	// a few convenience variables
	struct Config* p = &transformer->config;
	struct Weights* w = &transformer->weights;
	struct RunState* s = &transformer->state;
	float* x = s->x;
	int dim = p->dim;
	int kv_dim = (p->dim * p->n_kv_heads) / p->n_heads;
	int kv_mul = p->n_heads / p->n_kv_heads; // integer multiplier of the kv sharing in multiquery
	int hidden_dim = p->hidden_dim;
	int head_size = dim / p->n_heads;

	// ensure all dimensions are warp-aligned
	assert(dim % 32 == 0 && kv_dim % 32 == 0 && hidden_dim % 32 == 0);
	assert(p->vocab_size % 32 == 0);

	// rmsnorm and softmax require a larger-than-warp block size for efficiency
	const int rmsnorm_size = 1024;
	const int softmax_size = 1024;

	// copy the token embedding into x
	assert(token < p->vocab_size);
	kernel_embed<<<dim / 32, 32>>>(x, w->token_embedding_table + token * dim, dim);
	profiler_trigger("embed", 0);

	// forward all the layers
	for (int l = 0; l < p->n_layers; l++) {
		int loff = l * p->seq_len * kv_dim; // kv cache layer offset for convenience

		// attention rmsnorm
		kernel_rmsnorm<<<1, rmsnorm_size>>>(s->xb, x, w->rms_att_weight[l], dim);
		profiler_trigger("rmsnorm", 0);

		// qkv matmuls for this position
		kernel_matmul_qkv<<<dim + kv_dim * 2, 32>>>(s->q, s->k, s->v, s->xb, w->wq[l], w->wk[l], w->wv[l], dim, dim, kv_dim);
		profiler_trigger("matmul_qkv", (dim + kv_dim * 2) * dim * sizeof(dtype_t));

		// RoPE relative positional encoding: complex-valued rotate q and k in each head, and update kv cache
		assert(dim % 64 == 0 && kv_dim % 64 == 0);
		kernel_rope_kv<<<dim / 64, 32>>>(s->q, s->k, s->v, s->key_cache + loff, s->value_cache + loff, head_size, pos, p->rope_theta, dim, kv_dim, p->seq_len);
		profiler_trigger("rope_kv", 0);

		// only update kv cache and don't output logits
		if (l == p->n_layers - 1 && (flags & FF_UPDATE_KV_ONLY) != 0) {
			break;
		}

		// attention scores for all heads
		kernel_attn_score<<<dim3(pos + 1, p->n_kv_heads), dim3(32, kv_mul)>>>(s->att, s->q, s->key_cache + loff, p->n_kv_heads, head_size, p->seq_len, kv_dim, kv_mul, pos);
		profiler_trigger("attn_score", p->n_kv_heads * (pos + 1) * head_size * sizeof(kvtype_t));

		// softmax the scores to get attention weights, from 0..pos inclusively
		kernel_attn_softmax<<<p->n_heads, softmax_size>>>(s->att, p->n_heads, p->seq_len, pos);
		profiler_trigger("attn_softmax", 0);

		// compute weighted sum of the values into xb
		assert(head_size % 32 == 0);
		kernel_attn_mix<<<dim3(head_size, p->n_kv_heads), dim3(32, kv_mul)>>>(s->xb, s->att, s->value_cache + loff, p->n_kv_heads, head_size, p->seq_len, kv_dim, kv_mul, pos);
		profiler_trigger("attn_mix", p->n_kv_heads * (pos + 1) * head_size * sizeof(kvtype_t));

		// final matmul to get the output of the attention
		kernel_matmul_attn<<<dim, 32>>>(x, s->xb, w->wo[l], dim, dim);
		profiler_trigger("matmul_attn", dim * dim * sizeof(dtype_t));

		// ffn rmsnorm
		kernel_rmsnorm<<<1, rmsnorm_size>>>(s->xb, x, w->rms_ffn_weight[l], dim);
		profiler_trigger("rmsnorm", 0);

		// self.w2(F.silu(self.w1(x)) * self.w3(x)) + pre-rmsnorm residual
		kernel_matmul_ffn13<<<hidden_dim, 32>>>(s->hb, s->xb, w->w1[l], w->w3[l], dim, hidden_dim);
		profiler_trigger("matmul_ffn13", 2 * hidden_dim * dim * sizeof(dtype_t));

		kernel_matmul_ffn2<<<dim, 32>>>(x, s->hb, w->w2[l], hidden_dim, dim);
		profiler_trigger("matmul_ffn2", dim * hidden_dim * sizeof(dtype_t));
	}

	if (flags & FF_UPDATE_KV_ONLY) {
		// only update kv cache and don't output logits
		profiler_endsync();

		return NULL;
	}

	// final rmsnorm
	kernel_rmsnorm<<<1, rmsnorm_size>>>(x, x, w->rms_final_weight, dim);
	profiler_trigger("rmsnorm", 0);

	// classifier into logits
	kernel_matmul_cls<<<p->vocab_size, 32>>>(s->logits, x, w->wcls, dim, p->vocab_size);
	profiler_trigger("matmul_cls", p->vocab_size * dim * sizeof(dtype_t));

	profiler_endsync();

	CUDA_SYNC();

	return s->logits;
}
