#include "hip/hip_runtime.h"
#include "model.h"

#include "profiler.h"

#include <assert.h>
#include <math.h>
#include <stdio.h>

#include "helpers.cuh"

#define CUDA_CHECK(x)                                                                                    \
	do {                                                                                                 \
		hipError_t err = x;                                                                             \
		if (err != hipSuccess) {                                                                        \
			fprintf(stderr, "CUDA error in %s at %s:%d: %s (%s=%d)\n", __FUNCTION__, __FILE__, __LINE__, \
			        hipGetErrorString(err), hipGetErrorName(err), err);                                \
			abort();                                                                                     \
		}                                                                                                \
	} while (0)

#define CUDA_SYNC() CUDA_CHECK(hipDeviceSynchronize())

static void* cuda_devicecopy(void* host, size_t size) {
	void* device = NULL;
	CUDA_CHECK(hipMalloc(&device, size));
	CUDA_CHECK(hipMemcpy(device, host, size, hipMemcpyHostToDevice));
	return device;
}

static void* cuda_devicealloc(size_t size) {
	void* ptr = NULL;
	CUDA_CHECK(hipMalloc(&ptr, size));
	return ptr;
}

static void* cuda_hostalloc(size_t size) {
	void* ptr = NULL;
	CUDA_CHECK(hipHostAlloc(&ptr, size, 0));
	return ptr;
}

extern "C" void prepare_cuda(struct Transformer* transformer) {
	struct Config* config = &transformer->config;
	struct Weights* weights = &transformer->weights;
	struct RunState* state = &transformer->state;

	hipDeviceProp_t devprop = {};
	CUDA_CHECK(hipGetDeviceProperties(&devprop, 0));

	printf("# CUDA: %s, compute %d.%d, %d SMs, %.1f GiB, peak bandwidth %.0f GB/s\n",
	       devprop.name, devprop.major, devprop.minor, devprop.multiProcessorCount,
	       (double)devprop.totalGlobalMem / (1024 * 1024 * 1024),
	       (double)devprop.memoryClockRate * (devprop.memoryBusWidth / 8) * 2 / 1e6);

	int dim = config->dim;
	int hidden_dim = config->hidden_dim;
	int kv_dim = (config->dim * config->n_kv_heads) / config->n_heads;

	for (int l = 0; l < config->n_layers; ++l) {
		weights->rms_att_weight[l] = (float*)cuda_devicecopy(weights->rms_att_weight[l], dim * sizeof(float));
		weights->rms_ffn_weight[l] = (float*)cuda_devicecopy(weights->rms_ffn_weight[l], dim * sizeof(float));

		weights->wq[l] = cuda_devicecopy(weights->wq[l], dim * dim * weights->dsize);
		weights->wk[l] = cuda_devicecopy(weights->wk[l], dim * kv_dim * weights->dsize);
		weights->wv[l] = cuda_devicecopy(weights->wv[l], dim * kv_dim * weights->dsize);
		weights->wo[l] = cuda_devicecopy(weights->wo[l], dim * dim * weights->dsize);

		weights->w1[l] = cuda_devicecopy(weights->w1[l], dim * hidden_dim * weights->dsize);
		weights->w2[l] = cuda_devicecopy(weights->w2[l], dim * hidden_dim * weights->dsize);
		weights->w3[l] = cuda_devicecopy(weights->w3[l], dim * hidden_dim * weights->dsize);
	}

	weights->rms_final_weight = (float*)cuda_devicecopy(weights->rms_final_weight, dim * sizeof(float));
	weights->token_embedding_table = cuda_devicecopy(weights->token_embedding_table, config->vocab_size * dim * weights->dsize);
	weights->wcls = cuda_devicecopy(weights->wcls, dim * config->vocab_size * weights->dsize);

	state->x = (float*)cuda_devicealloc(dim * sizeof(float));
	state->xb = (float*)cuda_devicealloc(dim * sizeof(float));
	state->hb = (float*)cuda_devicealloc(hidden_dim * sizeof(float));
	state->q = (float*)cuda_devicealloc(dim * sizeof(float));
	state->k = (float*)cuda_devicealloc(kv_dim * sizeof(float));
	state->v = (float*)cuda_devicealloc(kv_dim * sizeof(float));
	state->att = (float*)cuda_devicealloc(config->n_heads * config->seq_len * sizeof(float));

	state->key_cache = (kvtype_t*)cuda_devicealloc((size_t)config->n_layers * config->seq_len * kv_dim * sizeof(kvtype_t));
	state->value_cache = (kvtype_t*)cuda_devicealloc((size_t)config->n_layers * config->seq_len * kv_dim * sizeof(kvtype_t));

	// logits are going to be read by the host so we just allocate them in host and write to host directly
	state->logits = (float*)cuda_hostalloc(config->vocab_size * sizeof(float));
}

template <typename T>
__global__ static void kernel_embed(float* o, T* weight, int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	assert(i < size);

	o[i] = float(weight[i]);
}

__global__ static void kernel_rmsnorm(float* o, float* x, float* weight, int size) {
	int i = threadIdx.x;
	int blockSize = blockDim.x;

	extern __shared__ float xs[];

	// calculate sum of squares (per thread)
	float ss = 0.0f;
	for (int j = i; j < size; j += blockSize) {
		float v = x[j];
		ss += v * v;

		// premultiply x by weight into shared memory to accelerate the second loop
		xs[j] = v * weight[j];
	}

	// sum across threads in block
	ss = blockreduce_sum(ss);

	// compute scale
	ss /= size;
	ss += 1e-5f;
	ss = 1.0f / sqrtf(ss);

	// normalize and scale
	// note: blockreduce above implies __syncthreads so xs[] reads are safe
	for (int j = i; j < size; j += blockSize) {
		o[j] = xs[j] * ss;
	}
}

template <typename T>
__global__ static void kernel_matmul_cls(float* xout, float* x, T* w, int n, int d) {
	int i = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
	assert(i < d);

	float val = matmul_warppar(x, w, i, n, n);

	// instead of writing one value per block, we transpose the values and write all results from first warp
	val = blocktranspose(val, 0.f);

	if (threadIdx.x < blockDim.x / warpSize) {
		xout[i + threadIdx.x] = val;
	}
}

template <typename T>
__global__ static void kernel_matmul_qkv(float* qout, float* kout, float* vout, float* x, T* wq, T* wk, T* wv, int n, int d, int kvd) {
	int i = blockIdx.x;
	assert(i < d + kvd * 2);

	float* out = i < d ? qout : (i < d + kvd ? kout : vout);
	T* w = i < d ? wq : (i < d + kvd ? wk : wv);
	int j = i < d ? i : (i < d + kvd ? i - d : i - d - kvd);

	float val = matmul_warppar(x, w, j, n, n);
	if (threadIdx.x == 0) {
		out[j] = val;
	}
}

template <typename T>
__global__ static void kernel_matmul_attn(float* xout, float* x, T* w, int n, int d) {
	int i = blockIdx.x;
	assert(i < d);

	float val = matmul_warppar(x, w, i, n, n);

	if (threadIdx.x == 0) {
		// += for residual
		xout[i] += val;
	}
}

template <typename T>
__global__ static void kernel_matmul_ffn13(float* xout, float* x, T* w1, T* w3, int n, int d) {
	int i = blockIdx.x;
	assert(i < d);

	float v1 = matmul_warppar(x, w1, i, n, n);
	float v3 = matmul_warppar(x, w3, i, n, n);

	// silu(x)=x*σ(x), where σ(x) is the logistic sigmoid
	float val = v1;
	val *= 1.0f / (1.0f + expf(-v1));
	val *= v3;

	if (threadIdx.x == 0) {
		xout[i] = val;
	}
}

template <typename T>
__global__ static void kernel_matmul_ffn2(float* xout, float* x, T* w, int n, int d) {
	int i = blockIdx.x;
	assert(i < d);

	float val = matmul_warppar(x, w, i, n, n);

	if (threadIdx.x == 0) {
		// += for residual
		xout[i] += val;
	}
}

__global__ static void kernel_rope_qkv(float* q, float* k, float* v, kvtype_t* kb, kvtype_t* vb, int head_size, int pos, int kv_pos, int kv_sink, float theta_log2, int d, int kvd, int seq_len) {
	int i = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
	assert(i < d + kvd + kv_sink * kvd);

	int j = i < d ? i : (i < d + kvd ? i - d : i - d - kvd);

	int head_dim = j & (head_size - 1);
	float freq = exp2f(-theta_log2 * (head_dim / (float)head_size));
	float fcr, fci;
	sincosf(pos * freq, &fci, &fcr);

	if (i < d) {
		float q0 = q[j];
		float q1 = q[j + 1];
		float rq0 = q0 * fcr - q1 * fci;
		float rq1 = q0 * fci + q1 * fcr;

		q[j] = rq0;
		q[j + 1] = rq1;
	} else if (i < d + kvd) {
		float k0 = k[j];
		float k1 = k[j + 1];
		float rk0 = k0 * fcr - k1 * fci;
		float rk1 = k0 * fci + k1 * fcr;

		float v0 = v[j];
		float v1 = v[j + 1];

		k[j] = rk0;
		k[j + 1] = rk1;

		// update kvcache key/value
		kb[kv_pos * kvd + j] = rk0;
		kb[kv_pos * kvd + j + 1] = rk1;

		// note: v layout is transposed (we store all positions for a given head contiguously) to improve attn_mix performance
		vb[kv_pos + seq_len * j] = v0;
		vb[kv_pos + seq_len * (j + 1)] = v1;
	} else {
		// rotate sink tokens forward to keep pace with non-sink tokens
		float k0 = kb[j];
		float k1 = kb[j + 1];

		sincosf(freq, &fci, &fcr);

		float rk0 = k0 * fcr - k1 * fci;
		float rk1 = k0 * fci + k1 * fcr;

		kb[j] = rk0;
		kb[j + 1] = rk1;
	}
}

__global__ static void kernel_attn_score(float* attb, float* qb, kvtype_t* kb, int n_kv_heads, int head_size, int seq_len, int kv_dim, int kv_mul, int kv_len) {
	int t = blockIdx.x;
	assert(t < kv_len);

	int kvh = blockIdx.y;
	assert(kvh < n_kv_heads);

	int h = kvh * kv_mul + threadIdx.y;

	float* q = qb + h * head_size;
	kvtype_t* k = kb + t * kv_dim + kvh * head_size;
	float* att = attb + h * seq_len;

	float score = 0.0f;
	for (int j = threadIdx.x * 2; j < head_size; j += warpSize * 2) {
		float2 kk = __half22float2(*((half2*)&k[j]));
		float2 qq = *(float2*)&q[j];
		score += kk.x * qq.x;
		score += kk.y * qq.y;
	}

	score = warpreduce_sum(score);
	score /= sqrtf(head_size);

	if (threadIdx.x == 0) {
		att[t] = score;
	}
}

__global__ static void kernel_attn_softmax(float* attb, int n_heads, int seq_len, int kv_len) {
	int i = threadIdx.x;

	int h = blockIdx.x;
	assert(h < n_heads);

	float* att = attb + h * seq_len;

	// find max value per thread (for numerical stability)
	float max_val = -FLT_MAX;
	for (int j = i; j < kv_len; j += blockDim.x) {
		max_val = max(max_val, att[j]);
	}

	// max across threads in block
	max_val = blockreduce_max(max_val);

	// exp and sum per thread
	float sum = 0.0f;
	for (int j = i; j < kv_len; j += blockDim.x) {
		sum += expf(att[j] - max_val);
	}

	// sum across threads in block
	sum = blockreduce_sum(sum);

	// output normalized values
	for (int j = i; j < kv_len; j += blockDim.x) {
		att[j] = expf(att[j] - max_val) / sum;
	}
}

__global__ static void kernel_attn_mix(float* xout, float* attb, kvtype_t* valb, int n_kv_heads, int head_size, int seq_len, int kv_dim, int kv_mul, int kv_len) {
	int i = blockIdx.x;
	assert(i < head_size);

	int kvh = blockIdx.y;
	assert(kvh < n_kv_heads);

	int h = kvh * kv_mul + threadIdx.y;

	float* att = attb + h * seq_len;
	kvtype_t* val = valb + (kvh * head_size + i) * seq_len;

	float res = 0.0f;
	for (int t = threadIdx.x * 2; t + 1 < kv_len; t += warpSize * 2) {
		float2 vv = __half22float2(*((half2*)&val[t]));
		float2 aa = *(float2*)&att[t];
		res += vv.x * aa.x;
		res += vv.y * aa.y;
	}

	if (kv_len % 2 == 1 && threadIdx.x == 0) {
		res += att[kv_len - 1] * float(val[kv_len - 1]);
	}

	res = warpreduce_sum(res);

	if (threadIdx.x == 0) {
		xout[h * head_size + i] = res;
	}
}

template <typename T>
static float* forward(struct Transformer* transformer, int token, int pos, unsigned flags) {
	profiler_begin();

	// a few convenience variables
	struct Config* p = &transformer->config;
	struct Weights* w = &transformer->weights;
	struct RunState* s = &transformer->state;
	float* x = s->x;
	int dim = p->dim;
	int kv_dim = (p->dim * p->n_kv_heads) / p->n_heads;
	int kv_mul = p->n_heads / p->n_kv_heads; // integer multiplier of the kv sharing in multiquery
	int hidden_dim = p->hidden_dim;
	int head_size = dim / p->n_heads;

	// following "attention sinks" from StreamingLLM we keep the first few tokens in the KV cache as is
	int kv_sink = pos >= p->seq_len ? KV_SINKS : 0;
	int kv_pos = kv_sink + (pos - kv_sink) % (p->seq_len - kv_sink);
	int kv_len = pos >= p->seq_len ? p->seq_len : pos + 1;

	// ensure all dimensions are warp-aligned
	assert(dim % 32 == 0 && kv_dim % 32 == 0 && hidden_dim % 32 == 0);
	assert(p->vocab_size % 32 == 0);

	// rmsnorm and softmax require a larger-than-warp block size for efficiency
	const int rmsnorm_size = 1024;
	const int softmax_size = 1024;

	// copy the token embedding into x
	assert(token < p->vocab_size);
	kernel_embed<<<dim / 32, 32>>>(x, (T*)w->token_embedding_table + token * dim, dim);
	profiler_trigger("embed", 0);

	// forward all the layers
	for (int l = 0; l < p->n_layers; l++) {
		int loff = l * p->seq_len * kv_dim; // kv cache layer offset for convenience

		// attention rmsnorm
		kernel_rmsnorm<<<1, rmsnorm_size, dim * sizeof(float)>>>(s->xb, x, w->rms_att_weight[l], dim);
		profiler_trigger("rmsnorm", 0);

		// qkv matmuls for this position
		kernel_matmul_qkv<<<dim + kv_dim * 2, 32>>>(s->q, s->k, s->v, s->xb, (T*)w->wq[l], (T*)w->wk[l], (T*)w->wv[l], dim, dim, kv_dim);
		profiler_trigger("matmul_qkv", (dim + kv_dim * 2) * dim * sizeof(T));

		// RoPE relative positional encoding: complex-valued rotate q and k in each head, and update kv cache
		assert(dim % 64 == 0 && kv_dim % 64 == 0);
		assert((head_size & (head_size - 1)) == 0); // head_size must be a power of 2
		kernel_rope_qkv<<<(dim + kv_dim + kv_dim * kv_sink) / 64, 32>>>(s->q, s->k, s->v, s->key_cache + loff, s->value_cache + loff, head_size, pos, kv_pos, kv_sink, log2(p->rope_theta), dim, kv_dim, p->seq_len);
		profiler_trigger("rope_qkv", 0);

		// only update kv cache and don't output logits
		if (l == p->n_layers - 1 && (flags & FF_UPDATE_KV_ONLY) != 0) {
			break;
		}

		// attention scores for all heads
		kernel_attn_score<<<dim3(kv_len, p->n_kv_heads), dim3(32, kv_mul)>>>(s->att, s->q, s->key_cache + loff, p->n_kv_heads, head_size, p->seq_len, kv_dim, kv_mul, kv_len);
		profiler_trigger("attn_score", p->n_kv_heads * kv_len * head_size * sizeof(kvtype_t));

		// softmax the scores to get attention weights over [0..kv_len)
		kernel_attn_softmax<<<p->n_heads, softmax_size>>>(s->att, p->n_heads, p->seq_len, kv_len);
		profiler_trigger("attn_softmax", 0);

		// compute weighted sum of the values into xb
		assert(head_size % 32 == 0);
		kernel_attn_mix<<<dim3(head_size, p->n_kv_heads), dim3(32, kv_mul)>>>(s->xb, s->att, s->value_cache + loff, p->n_kv_heads, head_size, p->seq_len, kv_dim, kv_mul, kv_len);
		profiler_trigger("attn_mix", p->n_kv_heads * kv_len * head_size * sizeof(kvtype_t));

		// final matmul to get the output of the attention
		kernel_matmul_attn<<<dim, 32>>>(x, s->xb, (T*)w->wo[l], dim, dim);
		profiler_trigger("matmul_attn", dim * dim * sizeof(T));

		// ffn rmsnorm
		kernel_rmsnorm<<<1, rmsnorm_size, dim * sizeof(float)>>>(s->xb, x, w->rms_ffn_weight[l], dim);
		profiler_trigger("rmsnorm", 0);

		// self.w2(F.silu(self.w1(x)) * self.w3(x)) + pre-rmsnorm residual
		kernel_matmul_ffn13<<<hidden_dim, 32>>>(s->hb, s->xb, (T*)w->w1[l], (T*)w->w3[l], dim, hidden_dim);
		profiler_trigger("matmul_ffn13", 2 * hidden_dim * dim * sizeof(T));

		kernel_matmul_ffn2<<<dim, 32>>>(x, s->hb, (T*)w->w2[l], hidden_dim, dim);
		profiler_trigger("matmul_ffn2", dim * hidden_dim * sizeof(T));
	}

	if (flags & FF_UPDATE_KV_ONLY) {
		// only update kv cache and don't output logits
		profiler_endsync();

		return NULL;
	}

	// final rmsnorm
	kernel_rmsnorm<<<1, rmsnorm_size, dim * sizeof(float)>>>(x, x, w->rms_final_weight, dim);
	profiler_trigger("rmsnorm", 0);

	// classifier into logits
	kernel_matmul_cls<<<p->vocab_size / 32, 32 * 32>>>(s->logits, x, (T*)w->wcls, dim, p->vocab_size);
	profiler_trigger("matmul_cls", p->vocab_size * dim * sizeof(T));

	profiler_endsync();

	CUDA_SYNC();

	return s->logits;
}

extern "C" float* forward_cuda(struct Transformer* transformer, int token, int pos, unsigned flags) {
	switch (transformer->weights.dsize) {
	case 1:
		return forward<__hip_fp8_e5m2_fnuz>(transformer, token, pos, flags);
	case 2:
		return forward<half>(transformer, token, pos, flags);
	default:
		return NULL;
	}
}
