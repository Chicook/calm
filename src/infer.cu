#include "hip/hip_runtime.h"
#include "model.h"

#include <assert.h>
#include <math.h>
#include <stdio.h>

#include <hip/hip_fp16.h>

#include "helpers.cuh"

typedef __half cudtype_t;

#define CUDA_CHECK(x)                                                                                    \
	do {                                                                                                 \
		hipError_t err = x;                                                                             \
		if (err != hipSuccess) {                                                                        \
			fprintf(stderr, "CUDA error in %s at %s:%d: %s (%s=%d)\n", __FUNCTION__, __FILE__, __LINE__, \
			        hipGetErrorString(err), hipGetErrorName(err), err);                                \
			abort();                                                                                     \
		}                                                                                                \
	} while (0)

#define CUDA_SYNC() CUDA_CHECK(hipDeviceSynchronize())

static void* cuda_devicecopy(void* host, size_t size) {
	void* device = NULL;
	CUDA_CHECK(hipMalloc(&device, size));
	CUDA_CHECK(hipMemcpy(device, host, size, hipMemcpyHostToDevice));
	return device;
}

static void* cuda_devicealloc(size_t size) {
	void* ptr = NULL;
	CUDA_CHECK(hipMalloc(&ptr, size));
	return ptr;
}

static void* cuda_hostalloc(size_t size) {
	void* ptr = NULL;
	CUDA_CHECK(hipHostAlloc(&ptr, size, 0));
	return ptr;
}

extern "C" void prepare_cuda(struct Transformer* transformer) {
	struct Config* config = &transformer->config;
	struct Weights* weights = &transformer->weights;
	struct RunState* state = &transformer->state;

	int dim = config->dim;
	int hidden_dim = config->hidden_dim;
	int kv_dim = (config->dim * config->n_kv_heads) / config->n_heads;

	for (int l = 0; l < config->n_layers; ++l) {
		weights->rms_att_weight[l] = (dtype_t*)cuda_devicecopy(weights->rms_att_weight[l], dim * sizeof(dtype_t));
		weights->rms_ffn_weight[l] = (dtype_t*)cuda_devicecopy(weights->rms_ffn_weight[l], dim * sizeof(dtype_t));

		weights->wq[l] = (dtype_t*)cuda_devicecopy(weights->wq[l], dim * dim * sizeof(dtype_t));
		weights->wk[l] = (dtype_t*)cuda_devicecopy(weights->wk[l], dim * kv_dim * sizeof(dtype_t));
		weights->wv[l] = (dtype_t*)cuda_devicecopy(weights->wv[l], dim * kv_dim * sizeof(dtype_t));
		weights->wo[l] = (dtype_t*)cuda_devicecopy(weights->wo[l], dim * dim * sizeof(dtype_t));

		weights->w1[l] = (dtype_t*)cuda_devicecopy(weights->w1[l], dim * hidden_dim * sizeof(dtype_t));
		weights->w2[l] = (dtype_t*)cuda_devicecopy(weights->w2[l], dim * hidden_dim * sizeof(dtype_t));
		weights->w3[l] = (dtype_t*)cuda_devicecopy(weights->w3[l], dim * hidden_dim * sizeof(dtype_t));
	}

	weights->rms_final_weight = (dtype_t*)cuda_devicecopy(weights->rms_final_weight, dim * sizeof(dtype_t));
	weights->token_embedding_table = (dtype_t*)cuda_devicecopy(weights->token_embedding_table, config->vocab_size * dim * sizeof(dtype_t));
	weights->wcls = (dtype_t*)cuda_devicecopy(weights->wcls, dim * config->vocab_size * sizeof(dtype_t));

	state->x = (float*)cuda_devicealloc(dim * sizeof(float));
	state->xb = (float*)cuda_devicealloc(dim * sizeof(float));
	state->hb = (float*)cuda_devicealloc(hidden_dim * sizeof(float));
	state->q = (float*)cuda_devicealloc(dim * sizeof(float));
	state->key_cache = (float*)cuda_devicealloc(config->n_layers * config->seq_len * kv_dim * sizeof(float));
	state->value_cache = (float*)cuda_devicealloc(config->n_layers * config->seq_len * kv_dim * sizeof(float));
	state->att = (float*)cuda_devicealloc(config->n_heads * config->seq_len * sizeof(float));

	// logits are going to be read by the host so we just allocate them in host and write to host directly
	state->logits = (float*)cuda_hostalloc(config->vocab_size * sizeof(float));
}

__global__ static void kernel_embed(float* o, cudtype_t* weight, int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	assert(i < size);

	o[i] = float(weight[i]);
}

__global__ static void kernel_rmsnorm(float* o, float* x, cudtype_t* weight, int size) {
	int i = threadIdx.x;
	int blockSize = blockDim.x;

	// calculate sum of squares (per thread)
	float ss = 0.0f;
	for (int j = i; j < size; j += blockSize) {
		ss += x[j] * x[j];
	}

	// sum across threads in warp
	ss = warpreduce_sum(ss);

	// sum across warps in block
	assert(blockSize <= 32 * warpSize);
	int lane = i % warpSize;
	int warp = i / warpSize;

	__shared__ float ssb[32];
	ssb[warp] = ss;
	__syncthreads();
	ss = warpreduce_sum(ssb[lane]);

	// compute scale
	ss /= size;
	ss += 1e-5f;
	ss = 1.0f / sqrtf(ss);

	// normalize and scale
	for (int j = i; j < size; j += blockSize) {
		o[j] = float(weight[j]) * (ss * x[j]);
	}
}

__global__ static void kernel_matmul_cls(float* xout, float* x, cudtype_t* w, int n, int d) {
	int i = blockIdx.x;
	assert(i < d);

	float val = matmul_warppar(x, w, i, n);

	if (threadIdx.x == 0) {
		xout[i] = val;
	}
}

__global__ static void kernel_matmul_qkv(float* qout, float* kout, float* vout, float* x, cudtype_t* wq, cudtype_t* wk, cudtype_t* wv, int n, int d, int kvd) {
	int i = blockIdx.x;
	assert(i < d + kvd * 2);

	float* out = i < d ? qout : (i < d + kvd ? kout : vout);
	cudtype_t* w = i < d ? wq : (i < d + kvd ? wk : wv);
	int j = i < d ? i : (i < d + kvd ? i - d : i - d - kvd);

	float val = matmul_warppar(x, w, j, n);
	if (threadIdx.x == 0) {
		out[j] = val;
	}
}

__global__ static void kernel_matmul_attn(float* xout, float* x, cudtype_t* w, int n, int d) {
	int i = blockIdx.x;
	assert(i < d);

	float val = matmul_warppar(x, w, i, n);

	if (threadIdx.x == 0) {
		// += for residual
		xout[i] += val;
	}
}

__global__ static void kernel_matmul_ffn13(float* xout, float* x, cudtype_t* w1, cudtype_t* w3, int n, int d) {
	int i = blockIdx.x;
	assert(i < d);

	float v1 = matmul_warppar(x, w1, i, n);
	float v3 = matmul_warppar(x, w3, i, n);

	// silu(x)=x*σ(x), where σ(x) is the logistic sigmoid
	float val = v1;
	val *= 1.0f / (1.0f + expf(-v1));
	val *= v3;

	if (threadIdx.x == 0) {
		xout[i] = val;
	}
}

__global__ static void kernel_matmul_ffn2(float* xout, float* x, cudtype_t* w, int n, int d) {
	int i = blockIdx.x;
	assert(i < d);

	float val = matmul_warppar(x, w, i, n);

	if (threadIdx.x == 0) {
		// += for residual
		xout[i] += val;
	}
}

__global__ static void kernel_rope(float* q, float* k, int head_size, int pos, float theta, int d, int kvd) {
	int i = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
	assert(i < d);

	int head_dim = i % head_size;
	float freq = 1.0f / powf(theta, head_dim / (float)head_size);
	float val = pos * freq;
	float fcr = cosf(val);
	float fci = sinf(val);

	float q0 = q[i];
	float q1 = q[i + 1];
	q[i] = q0 * fcr - q1 * fci;
	q[i + 1] = q0 * fci + q1 * fcr;

	if (i < kvd) {
		float k0 = k[i];
		float k1 = k[i + 1];
		k[i] = k0 * fcr - k1 * fci;
		k[i + 1] = k0 * fci + k1 * fcr;
	}
}

__global__ static void kernel_attn_score(float* attb, float* qb, float* kb, int n_heads, int head_size, int seq_len, int kv_dim, int kv_mul, int pos) {
	int t = blockIdx.x * blockDim.x + threadIdx.x;
	if (t > pos) {
		return;
	}

	int h = blockIdx.y;
	assert(h < n_heads);

	float* q = qb + h * head_size;
	float* k = kb + t * kv_dim + (h / kv_mul) * head_size;
	float* att = attb + h * seq_len;

	float score = 0.0f;
	for (int j = 0; j < head_size; j++) {
		score += q[j] * k[j];
	}
	score /= sqrtf(head_size);

	att[t] = score;
}

__global__ static void kernel_attn_softmax(float* attb, int n_heads, int seq_len, int pos) {
	int i = threadIdx.x;

	int h = blockIdx.y;
	assert(h < n_heads);

	float* att = attb + h * seq_len;

	// find max value per thread (for numerical stability)
	float max_val = 0.f;
	for (int j = i; j <= pos; j += warpSize) {
		max_val = max(max_val, att[j]);
	}

	// max across threads
	max_val = warpreduce_max(max_val);

	// exp and sum per thread
	float sum = 0.0f;
	for (int j = i; j <= pos; j += warpSize) {
		sum += expf(att[j] - max_val);
	}

	// sum across threads
	sum = warpreduce_sum(sum);

	// output normalized values
	for (int j = i; j <= pos; j += warpSize) {
		att[j] = expf(att[j] - max_val) / sum;
	}
}

__global__ static void kernel_attn_mix(float* xout, float* attb, float* valb, int n_heads, int head_size, int seq_len, int kv_dim, int kv_mul, int pos) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	assert(i < head_size);

	int h = blockIdx.y;
	assert(h < n_heads);

	float* att = attb + h * seq_len;
	float* val = valb + (h / kv_mul) * head_size;

	float res = 0.0f;
	for (int t = 0; t <= pos; t++) {
		res += att[t] * val[t * kv_dim + i];
	}

	xout[h * head_size + i] = res;
}

extern "C" float* forward_cuda(struct Transformer* transformer, int token, int pos) {

	// a few convenience variables
	struct Config* p = &transformer->config;
	struct Weights* w = &transformer->weights;
	struct RunState* s = &transformer->state;
	float* x = s->x;
	int dim = p->dim;
	int kv_dim = (p->dim * p->n_kv_heads) / p->n_heads;
	int kv_mul = p->n_heads / p->n_kv_heads; // integer multiplier of the kv sharing in multiquery
	int hidden_dim = p->hidden_dim;
	int head_size = dim / p->n_heads;

	// ensure all dimensions are warp-aligned
	assert(dim % 32 == 0 && kv_dim % 32 == 0 && hidden_dim % 32 == 0);
	assert(p->vocab_size % 32 == 0);

	// rmsnorm requires a larger-than-warp block size for efficiency
	const int rmsnorm_size = 1024;
	assert(dim % rmsnorm_size == 0);

	// copy the token embedding into x
	assert(token < p->vocab_size);
	kernel_embed<<<dim / 32, 32>>>(x, (cudtype_t*)w->token_embedding_table + token * dim, dim);

	// forward all the layers
	for (int l = 0; l < p->n_layers; l++) {

		// attention rmsnorm
		kernel_rmsnorm<<<1, rmsnorm_size>>>(s->xb, x, (cudtype_t*)w->rms_att_weight[l], dim);

		// key and value point to the kv cache
		int loff = l * p->seq_len * kv_dim; // kv cache layer offset for convenience
		s->k = s->key_cache + loff + pos * kv_dim;
		s->v = s->value_cache + loff + pos * kv_dim;

		// qkv matmuls for this position
		kernel_matmul_qkv<<<dim + kv_dim * 2, 32>>>(s->q, s->k, s->v, s->xb, (cudtype_t*)w->wq[l], (cudtype_t*)w->wk[l], (cudtype_t*)w->wv[l], dim, dim, kv_dim);

		// RoPE relative positional encoding: complex-valued rotate q and k in each head
		assert(dim % 64 == 0 && kv_dim % 64 == 0);
		kernel_rope<<<dim / 64, 32>>>(s->q, s->k, head_size, pos, p->rope_theta, dim, kv_dim);

		// attention scores for all heads
		kernel_attn_score<<<dim3((pos + 1 + 31) / 32, p->n_heads), 32>>>(s->att, s->q, s->key_cache + loff, p->n_heads, head_size, p->seq_len, kv_dim, kv_mul, pos);

		// softmax the scores to get attention weights, from 0..pos inclusively
		kernel_attn_softmax<<<dim3(1, p->n_heads), 32>>>(s->att, p->n_heads, p->seq_len, pos);

		// compute weighted sum of the values into xb
		assert(head_size % 32 == 0);
		kernel_attn_mix<<<dim3(head_size / 32, p->n_heads), 32>>>(s->xb, s->att, s->value_cache + loff, p->n_heads, head_size, p->seq_len, kv_dim, kv_mul, pos);

		// final matmul to get the output of the attention
		kernel_matmul_attn<<<dim, 32>>>(x, s->xb, (cudtype_t*)w->wo[l], dim, dim);

		// ffn rmsnorm
		kernel_rmsnorm<<<1, rmsnorm_size>>>(s->xb, x, (cudtype_t*)w->rms_ffn_weight[l], dim);

		// self.w2(F.silu(self.w1(x)) * self.w3(x)) + pre-rmsnorm residual
		kernel_matmul_ffn13<<<hidden_dim, 32>>>(s->hb, s->xb, (cudtype_t*)w->w1[l], (cudtype_t*)w->w3[l], dim, hidden_dim);
		kernel_matmul_ffn2<<<dim, 32>>>(x, s->hb, (cudtype_t*)w->w2[l], hidden_dim, dim);
	}

	// final rmsnorm
	kernel_rmsnorm<<<1, rmsnorm_size>>>(x, x, (cudtype_t*)w->rms_final_weight, dim);

	// classifier into logits
	kernel_matmul_cls<<<p->vocab_size, 32>>>(s->logits, x, (cudtype_t*)w->wcls, p->dim, p->vocab_size);

	CUDA_SYNC();

	return s->logits;
}
